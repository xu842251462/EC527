
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <time.h>

#define OMEGA 1.90       // TO BE DETERMINED

typedef double data_t;

typedef struct {
    long int rowlen;
    data_t *data;
} arr_rec, *arr_ptr;


#define NUM_THREADS_PER_BLOCK   256
#define NUM_BLOCKS         16
#define PRINT_TIME         1
#define SM_ARR_LEN        (1<<11)
#define TOL            5e-2
#define IMUL(a, b) __mul24(a, b)
void initializeArray1D(float *arr, int len, int seed);
void SOR_blocked(arr_ptr v, int *iterations);

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

//kernal, find the starting index of every block in the matrix
__global__ void SOR_not_inter(float *v, int len) {
    float result = 0;
    int idx = blockIdx.x + threadIdx.x*gridDim.x;
    int idy = blockIdx.y + threadIdx.y*gridDim.y;
    int id  = idx*len+idy;
    if(id>len && id<(len-1)*len && (id%len!=0) && (id%len!=len-1)){
        result = v[id] - 0.25 * (v[id - 1] + v[id + 1] + v[id - len] + v[id + len]);
        v[id] -= result * 1.9;
    }

}

__global__ void SOR_inter(float *v, int len) {
    float result = 0;
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int idy = threadIdx.y + blockIdx.y*blockDim.y;
    int id  = idx*len+idy;
    if(id>len && id<(len-1)*len && (id%len!=0) && (id%len!=len-1)){
        result = v[id] - 0.25 * (v[id - 1] + v[id + 1] + v[id - len] + v[id + len]);
        v[id] -= result * 1.9;
    }

}

void SOR_cpu(float *v){
    data_t change;
    int k, i, j;
    int length = SM_ARR_LEN;
    for(k=0;k<2000;k++){
        for (i = 1; i < length-1; i++){
            for (j = 1; j < length-1; j++) {
                change = v[i*length+j] - .25 * (v[(i-1)*length+j] +
                                                  v[(i+1)*length+j] +
                                                  v[i*length+j+1] +
                                                  v[i*length+j-1]);
                v[i*length+j] -= change * OMEGA;
            }
        }
    }
}

int compare(float* h_result, float* h_result_gold){
    int i;
    int errCount =0;
    int zeroCount = 0;
    for(i = 0; i < SM_ARR_LEN*SM_ARR_LEN; i++) {
        if (abs(h_result_gold[i] - h_result[i]) > TOL*h_result_gold[i]) {
        errCount++;
        }
        if(h_result[i]==0)
        zeroCount++;
  }
  if (zeroCount>0)
    errCount = -1;
  return errCount;
}

/* -=-=-=-=- Time measurement by clock_gettime() -=-=-=-=- */
/*
  As described in the clock_gettime manpage (type "man clock_gettime" at the
  shell prompt), a "timespec" is a structure that looks like this:
 
        struct timespec {
          time_t   tv_sec;   // seconds
          long     tv_nsec;  // and nanoseconds
        };
 */

double interval(struct timespec start, struct timespec end)
{
  struct timespec temp;
  temp.tv_sec = end.tv_sec - start.tv_sec;
  temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  if (temp.tv_nsec < 0) {
    temp.tv_sec = temp.tv_sec - 1;
    temp.tv_nsec = temp.tv_nsec + 1000000000;
  }
  return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
}
/*
     This method does not require adjusting a #define constant

  How to use this method:

      struct timespec time_start, time_stop;
      clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_start);
      // DO SOMETHING THAT TAKES TIME
      clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_stop);
      measurement = interval(time_start, time_stop);

 */


/* This routine "wastes" a little time to make sure the machine gets
   out of power-saving mode (800 MHz) and switches to normal speed. */
double wakeup_delay()
{
  double meas = 0; int i, j;
  struct timespec time_start, time_stop;
  double quasi_random = 0;
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_start);
  j = 100;
  while (meas < 1.0) {
    for (i=1; i<j; i++) {
      /* This iterative calculation uses a chaotic map function, specifically
         the complex quadratic map (as in Julia and Mandelbrot sets), which is
         unpredictable enough to prevent compiler optimisation. */
      quasi_random = quasi_random*quasi_random - 1.923432;
    }
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_stop);
    meas = interval(time_start, time_stop);
    j *= 2; /* Twice as much delay next time, until we've taken 1 second */
  }
  return quasi_random;
}


int main(int argc, char **argv) {
    int arrLen = 0;
    float final_answer;
    final_answer = wakeup_delay();
    //GPU timing variables
    hipEvent_t start, stop;
    float elapsed_gpu;

    //arrays in GPU global memoryc
    float *d_x;

    //arrays on host memory
    float *h_x;
    float *h_y_cpu;
    float *h_y_gpu;

//    int i, errCount = 0, zeroCount = 0;

    if (argc > 1) {
        arrLen = atoi(argv[1]);
    } else {
        arrLen = SM_ARR_LEN * SM_ARR_LEN;
    }


    

    // Select GPU
    CUDA_SAFE_CALL(hipSetDevice(0));

    // Allocate GPU memory, d_x-destination
    size_t allocSize = arrLen * sizeof(float);
    CUDA_SAFE_CALL(hipMalloc((void **)&d_x, allocSize));
    // Allocate arrays on host memory
    h_x                        = (float *) malloc(allocSize);
    h_y_cpu                    = (float *) malloc(allocSize);
    h_y_gpu                    = (float *) malloc(allocSize);
    // Initialize the host arrays, h_x-source
    printf("\nInitializing the arrays ...");
    // Arrays are initialized with a known seed for reproducability
    initializeArray1D(h_x, arrLen, 2000);
    initializeArray1D(h_y_cpu, arrLen, 2000);
    initializeArray1D(h_y_gpu, arrLen, 2000);
    printf("\t... done\n\n");

    int i;
    for(i=0; i<SM_ARR_LEN*SM_ARR_LEN; i++){
            h_y_cpu[i] = h_x[i];
    }

    struct timespec diff(struct timespec start, struct timespec end);
    struct timespec time1, time2;
    struct timespec time_stamp;
#define GIG 5.0e9
    // Compute the results on the host
    printf("\ncalculating results on host: ");  

    clock_gettime(CLOCK_REALTIME, &time1);
    
    SOR_cpu(h_y_cpu);

    clock_gettime(CLOCK_REALTIME, &time2);
    time_stamp = diff(time1,time2);
    printf("%lf (msec)\n", ((double) (GIG * time_stamp.tv_sec + time_stamp.tv_nsec)/1000000));



#if PRINT_TIME
    // Create the cuda events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Record event on the default stream
    hipEventRecord(start, 0);
#endif

    // Transfer the arrays to the GPU memory
    CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, allocSize, hipMemcpyHostToDevice));

    //launch the kernel(function)
    dim3 dimBlock(16,16,1);
    dim3 dimGrid(SM_ARR_LEN/16,SM_ARR_LEN/16,1);
    for(int i=0; i<2000; i++)
    SOR_inter<<<dimGrid, dimBlock>>>(d_x, SM_ARR_LEN);

    //check for errors during launch
    CUDA_SAFE_CALL(hipPeekAtLastError());

    //transfer the results back to the host
    CUDA_SAFE_CALL(hipMemcpy(h_y_gpu, d_x, allocSize, hipMemcpyDeviceToHost));

#if PRINT_TIME
    // Stop and destroy the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    printf("\nGPU time block (interleaved): %f (msec)\n", elapsed_gpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);
#endif

printf("\nCompare: %d\n\n\n",compare(h_y_gpu,h_y_cpu));

#if PRINT_TIME
    // Create the cuda events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Record event on the default stream
    hipEventRecord(start, 0);
#endif

    // Transfer the arrays to the GPU memory
    CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, allocSize, hipMemcpyHostToDevice));

    //launch the kernel(function)
    dim3 dimBlockk(1,16*16,1);
    dim3 dimGridd(SM_ARR_LEN,SM_ARR_LEN/256,1);
    for(int i=0; i<2000; i++)
    SOR_inter<<<dimGridd, dimBlockk>>>(d_x, SM_ARR_LEN);

    //check for errors during launch
    CUDA_SAFE_CALL(hipPeekAtLastError());

    //transfer the results back to the host
    CUDA_SAFE_CALL(hipMemcpy(h_y_gpu, d_x, allocSize, hipMemcpyDeviceToHost));

#if PRINT_TIME
    // Stop and destroy the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    printf("\nGPU time strip (interleaved): %f (msec)\n", elapsed_gpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);
#endif

printf("\nCompare: %d\n\n\n",compare(h_y_gpu,h_y_cpu));

#if PRINT_TIME
    // Create the cuda events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Record event on the default stream
    hipEventRecord(start, 0);
#endif

    // Transfer the arrays to the GPU memory
    CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, allocSize, hipMemcpyHostToDevice));    

    //launch the kernel(function)
    //dim3 dimBlockk(1,16*16);
    //dim3 dimGridd(1<<11,8);
    for(int i=0; i<2000; i++)
    SOR_not_inter<<<dimGrid, dimBlock>>>(d_x, SM_ARR_LEN);

    //check for errors during launch
    CUDA_SAFE_CALL(hipPeekAtLastError());

    //transfer the results back to the host
    CUDA_SAFE_CALL(hipMemcpy(h_y_gpu, d_x, allocSize, hipMemcpyDeviceToHost));

#if PRINT_TIME
    // Stop and destroy the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    printf("\nGPU time block (not interleaved): %f (msec)\n", elapsed_gpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);
#endif

printf("\nCompare: %d\n\n\n",compare(h_y_gpu,h_y_cpu));


#if PRINT_TIME
    // Create the cuda events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Record event on the default stream
    hipEventRecord(start, 0);
#endif

    // Transfer the arrays to the GPU memory
    CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, allocSize, hipMemcpyHostToDevice));    

    //launch the kernel(function)
    //dim3 dimBlockk(1,16*16);
    //dim3 dimGridd(1<<11,8);
    for(int i=0; i<2000; i++)
    SOR_not_inter<<<dimGridd, dimBlockk>>>(d_x,SM_ARR_LEN);

    //check for errors during launch
    CUDA_SAFE_CALL(hipPeekAtLastError());

    //transfer the results back to the host
    CUDA_SAFE_CALL(hipMemcpy(h_y_gpu, d_x, allocSize, hipMemcpyDeviceToHost));

#if PRINT_TIME
    // Stop and destroy the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    printf("\nGPU time strip (not interleaved): %f (msec)\n", elapsed_gpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);
#endif

printf("\nCompare: %d\n\n\n",compare(h_y_cpu,h_y_gpu));


    //free device and host memory
    CUDA_SAFE_CALL(hipFree(d_x));

    free(h_x);
    free(h_y_cpu);
    free(h_y_gpu);

    printf("\n");
  printf("Initial delay was calculating: %g \n", final_answer);

    return 0;
}

void initializeArray1D(float *arr, int len, int seed) {
    int i;
    float randNum;
    srand(seed);

    for (i = 0; i < len; i++) {
        randNum = (float) rand();
        arr[i] = randNum;
    }
}

struct timespec diff(struct timespec start, struct timespec end)
{
    struct timespec temp;
    if ((end.tv_nsec-start.tv_nsec)<0) {
        temp.tv_sec = end.tv_sec-start.tv_sec-1;
        temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
    } else {
        temp.tv_sec = end.tv_sec-start.tv_sec;
        temp.tv_nsec = end.tv_nsec-start.tv_nsec;
    }
    return temp;
}
